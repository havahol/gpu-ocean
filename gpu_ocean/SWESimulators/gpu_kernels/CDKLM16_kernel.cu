#include "hip/hip_runtime.h"
/*
This software is part of GPU Ocean. 

Copyright (C) 2018, 2019 SINTEF Digital
Copyright (C) 2018, 2019 Norwegian Meteorological Institute

This CUDA kernel implements the CDKLM numerical scheme
for the shallow water equations, described in
A. Chertock, M. Dudzinski, A. Kurganov & M. Lukacova-Medvidova
Well-Balanced Schemes for the Shallow Water Equations with Coriolis Forces,
Numerische Mathematik 2016

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "common.cu"
#include "angle_texture.cu"


// KPSIMULATOR

//WARNING: Must match max_dt.cu and initBm_kernel.cu
//WARNING: This is error prone - as comparison with floating point numbers is not accurate
#define CDKLM_DRY_FLAG 1.0e-30f
#define CDKLM_DRY_EPS 1.0e-3f


/**
  * Decompose the north vector to x and y coordinates
  */
__device__
inline float2 getNorth(const int i, const int j,
                       const int nx_, const int ny_) {
    // Get the angle towards north from the y-axis
    const float s = (i-2.0f+0.5f) / (float) nx_;
    const float t = (j-2.0f+0.5f) / (float) ny_;
    const float angle = tex2D(angle_tex, s, t);
    
    // Decompose (code inspired from the cdklm_swe_2D kernel)
    return make_float2(sinf(angle), cosf(angle));
}

/**
  * Decompose the east vector to x and y coordinates
  */
__device__
inline float2 getEast(const int i, const int j,
                      const int nx_, const int ny_) {

    const float2 north = getNorth(i, j, nx_, ny_);
    return make_float2(north.y, -north.x);
}


__device__ float3 CDKLM16_F_func(const float3 Q, const float g) {
    float3 F;

    F.x = Q.x*Q.y;                        //h*u
    F.y = Q.x*Q.y*Q.y + 0.5f*g*Q.x*Q.x;   //h*u*u + 0.5f*g*h*h;
    F.z = Q.x*Q.y*Q.z;                    //h*u*v;

    return F;
}







/**
  * Note that the input vectors are (h, u, v), thus not the regular
  * (h, hu, hv). 
  * Note also that u and v are desingularized from the start.
  */
__device__ float3 CDKLM16_flux(float3 Qm, float3 Qp, const float g) {
    
    // Contribution from plus cell
    float3 Fp = make_float3(0.0f, 0.0f, 0.0f);
    float up = 0.0f;
    float cp = 0.0f;
    
    if (Qp.x > KPSIMULATOR_DEPTH_CUTOFF) {
        Fp = CDKLM16_F_func(Qp, g);
        up = Qp.y;         // u
        cp = sqrtf(g*Qp.x); // sqrt(g*h)
    }

    // Contribution from plus cell
    float3 Fm = make_float3(0.0f, 0.0f, 0.0f);
    float um = 0.0f;
    float cm = 0.0f;

    if (Qm.x > KPSIMULATOR_DEPTH_CUTOFF) {
        Fm = CDKLM16_F_func(Qm, g);
        um = Qm.y;         // u
        cm = sqrtf(g*Qm.x); // sqrt(g*h)
    }
    
    const float am = min(min(um-cm, up-cp), 0.0f); // largest negative wave speed
    const float ap = max(max(um+cm, up+cp), 0.0f); // largest positive wave speed

    // If symmetric Rieman fan, return zero flux
    if ( fabsf(ap - am) < KPSIMULATOR_FLUX_SLOPE_EPS ) {
        return make_float3(0.0f, 0.0f, 0.0f);
    }
    
    float3 F;

    F.x = ((ap*Fm.x - am*Fp.x) + ap*am*(Qp.x-Qm.x))/(ap-am);
    F.y = ((ap*Fm.y - am*Fp.y) + ap*am*(Fp.x-Fm.x))/(ap-am);
    F.z = (Qm.y + Qp.y > 0) ? Fm.z : Fp.z; //Upwinding to be consistent

    return F;
}



/**
  * Adjusting the slope of K_x, found in Qx[3], to avoid negative values for h on the faces,
  * in the case of dry cells
  */
__device__
void adjustSlopes_x(const int bx, const int by, 
                    const int nx_, const int ny_, const float dx_, const float dy_,
                    float R[3][block_height+4][block_width+4],
                    float Qx[3][block_height+2][block_width+2], // used as if Qx[3][block_height][block_width + 2]
                    float Hi[block_height+3][block_width+3],
                    const float g_, 
                    const float f_, const float beta_, 
                    const int& bc_east_, const int& bc_west_) {
    
    // Need K_x (Qx[2]), coriolis parameter (f, beta), eta (R[0]), v (R[2]), H (Hi), g, dx

    
    const int j = threadIdx.y; // values in Qx
    const int l = j + 2; // values in R
    const int H_j = j + 1; // values in Hi
    
    for (int i=threadIdx.x; i<block_width+2; i+=blockDim.x) {
        // i referes to values in Qx
        const int k = i + 1; // values in R
        const int H_i = i; // values in Hi

        // Reconstruct h at east and west faces
        const float eta = R[0][l][k];
        
        float v   = R[2][l][k];
        // Fix west boundary for reconstruction of eta (corresponding to Kx)
        if ((bc_west_ == 1) && (bx + k < 2    )) { v = -v; }
        // Fix east boundary for reconstruction of eta (corresponding to Kx)
        if ((bc_east_ == 1) && (bx + k > nx_+2)) { v = -v; }
        
        // Coriolis in this cell
        const float2 north = getNorth(bx+k, by+l, nx_, ny_);
        const float coriolis_f = f_ + beta_ * ((bx + k + 0.5f - 2.0f)*dx_*north.x + 
                                               (by + l + 0.5f - 2.0f)*dy_*north.y);
        
        const float dxfv = dx_*coriolis_f*v;
        
        const float H_west = 0.5f*(Hi[H_j][H_i  ] + Hi[H_j+1][H_i  ]);
        const float H_east = 0.5f*(Hi[H_j][H_i+1] + Hi[H_j+1][H_i+1]);
        
        const float h_west = eta + H_west - (Qx[2][j][i] + dxfv)/(2.0f*g_);
        const float h_east = eta + H_east + (Qx[2][j][i] + dxfv)/(2.0f*g_);
        
        // Adjust if negative water level
        Qx[2][j][i] = (h_west > 0) ? Qx[2][j][i] : -dxfv + 2.0f*g_*(eta + H_west);
        Qx[2][j][i] = (h_east > 0) ? Qx[2][j][i] : -dxfv - 2.0f*g_*(eta + H_east);
    }
}


/**
  * Adjusting the slope of L_y, found in Qx[3], to avoid negative values for h on the faces,
  * in the case of dry cells
  */
__device__
void adjustSlopes_y(const int bx, const int by, 
                    const int nx_, const int ny_, const float dx_, const float dy_,
                    float R[3][block_height+4][block_width+4],
                    float Qx[3][block_height+2][block_width+2], // used as if Qx[3][block_height+2][block_width]
                    float Hi[block_height+3][block_width+3],
                    const float g_, 
                    const float f_, const float beta_,
                    const int& bc_north_, const int& bc_south_) {
    
    // Need K_x (Qx[2]), coriolis parameter (f, beta), eta (R[0]), v (R[2]), H (Hi), g, dx

    
    const int i = threadIdx.x; // values in Qx
    const int k = i + 2; // values in R
    const int H_i = i + 1; // values in Hi
    
    for (int j=threadIdx.y; j<block_height+2; j+=blockDim.y) {
        // i referes to values in Qx
        const int l = j + 1; // values in R
        const int H_j = j; // values in Hi

        // Reconstruct h at east and west faces
        const float eta = R[0][l][k];
        
        float u   = R[1][l][k];
        // Fix south boundary for reconstruction of eta (corresponding to Ly)
        if ((bc_south_ == 1) && (by + l < 2    )) { u = -u; }
        // Fix north boundary for reconstruction of eta (corresponding to Ly)
        if ((bc_north_ == 1) && (by + l > ny_+2)) { u = -u; }
        
        // Coriolis in this cell
        const float2 north = getNorth(bx+k, by+l, nx_, ny_);
        const float coriolis_f = f_ + beta_ * ((bx + k + 0.5f - 2.0f)*dx_*north.x + 
                                               (by + l + 0.5f - 2.0f)*dy_*north.y);

        const float dyfu = dy_*coriolis_f*u;
        
        const float H_south = 0.5f*(Hi[H_j  ][H_i] + Hi[H_j  ][H_i+1]);
        const float H_north = 0.5f*(Hi[H_j+1][H_i] + Hi[H_j+1][H_i+1]);
        
        const float h_south = eta + H_south - (Qx[2][j][i] - dyfu)/(2.0f*g_);
        const float h_north = eta + H_north + (Qx[2][j][i] - dyfu)/(2.0f*g_);
        
        // Adjust if negative water level
        Qx[2][j][i] = (h_south > 0) ? Qx[2][j][i] : dyfu + 2.0f*g_*(eta + H_south);
        Qx[2][j][i] = (h_north > 0) ? Qx[2][j][i] : dyfu - 2.0f*g_*(eta + H_north);
    }
}




__device__
float3 computeFFaceFlux(const int i, const int j, const int bx, const int nx_,
                float R[3][block_height+4][block_width+4],
                float Qx[3][block_height+2][block_width+2],
                float Hi[block_height+3][block_width+3],
                const float g_, const float coriolis_fm, const float coriolis_fp, const float dx_,
                const int& bc_east_, const int& bc_west_,
                const float2 north) {
    const int l = j + 2; //Skip ghost cells (be consistent with reconstruction offsets)
    const int k = i + 1;

    // Skip ghost cells in the Hi buffer
    const int H_i = i+1;
    const int H_j = j+1;
    
    // (u, v) reconstructed at a cell interface from the right (p) and left (m)
    // Variables to reconstruct h from u, v, K, L
    const float eta_bar_p = R[0][l][k+1];
    const float eta_bar_m = R[0][l][k  ];
    const float up = R[1][l][k+1];
    const float um = R[1][l][k  ];
    float vp = R[2][l][k+1];
    float vm = R[2][l][k  ];
    
    //Check if dry: if so return zero flux
    if (eta_bar_p == CDKLM_DRY_FLAG || eta_bar_m == CDKLM_DRY_FLAG) {
        return make_float3(0.0f, 0.0f, 0.0f);
    }

    const float2 Rp = make_float2(up - 0.5f*Qx[0][j][i+1], vp - 0.5f*Qx[1][j][i+1]);
    const float2 Rm = make_float2(um + 0.5f*Qx[0][j][i  ], vm + 0.5f*Qx[1][j][i  ]);

    // H is RHx on the given face!
    const float H_face = 0.5f*( Hi[H_j][H_i] + Hi[H_j+1][H_i] );

    // Qx[2] is really dx*Kx
    const float Kx_p = Qx[2][j][i+1];
    const float Kx_m = Qx[2][j][i  ];
    
    // Fix west boundary for reconstruction of eta (corresponding to Kx)
    if ((bc_west_ == 1) && (bx + i + 2 == 2    )) { vm = -vm; }
    // Fix east boundary for reconstruction of eta (corresponding to Kx)
    if ((bc_east_ == 1) && (bx + i + 2 == nx_+2)) { vp = -vp; }
    
    //Reconstruct momentum along north
    const float vp_north = up*north.x + vp*north.y;
    const float vm_north = um*north.x + vm*north.y;
    
    // Reconstruct h
    const float hp = fmaxf(0.0f, eta_bar_p + H_face - (Kx_p + dx_*coriolis_fp*vp_north)/(2.0f*g_));
    const float hm = fmaxf(0.0f, eta_bar_m + H_face + (Kx_m + dx_*coriolis_fm*vm_north)/(2.0f*g_));

    // Our flux variables Q=(h, u, v)
    const float3 Qp = make_float3(hp, Rp.x, Rp.y);
    const float3 Qm = make_float3(hm, Rm.x, Rm.y);

    // Computed flux
    return CDKLM16_flux(Qm, Qp, g_);
}




__device__
float3 computeGFaceFlux(const int i, const int j, const int by, const int ny_,
                float R[3][block_height+4][block_width+4],
                float Qy[3][block_height+2][block_width+2],
                float Hi[block_height+3][block_width+3],
                const float g_, const float coriolis_fm, const float coriolis_fp, const float dy_,
                const int& bc_north_, const int& bc_south_,
                const float2 east) {
    const int l = j + 1;
    const int k = i + 2; //Skip ghost cells
    
    // Skip ghost cells in the Hi buffer
    const int H_i = i+1;
    const int H_j = j+1;
    
    // Q at interface from the right and left
    // Variables to reconstruct h from u, v, K, L
    const float eta_bar_p = R[0][l+1][k];
    const float eta_bar_m = R[0][l  ][k];
    float up = R[1][l+1][k];
    float um = R[1][l  ][k];
    const float vp = R[2][l+1][k];
    const float vm = R[2][l  ][k];

    //Check if dry: if so return zero flux
    if (eta_bar_p == CDKLM_DRY_FLAG || eta_bar_m == CDKLM_DRY_FLAG) {
        return make_float3(0.0f, 0.0f, 0.0f);
    }
    
    const float2 Rp = make_float2(up - 0.5f*Qy[0][j+1][i], vp - 0.5f*Qy[1][j+1][i]);
    const float2 Rm = make_float2(um + 0.5f*Qy[0][j  ][i], vm + 0.5f*Qy[1][j  ][i]);

    // H is RHx on the given face!
    const float H_face = 0.5f*( Hi[H_j][H_i] + Hi[H_j][H_i+1] );

    // Qy[2] is really dy*Ly
    const float Ly_p = Qy[2][j+1][i];
    const float Ly_m = Qy[2][j  ][i];

    // Fix south boundary for reconstruction of eta (corresponding to Ly)
    if ((bc_south_ == 1) && (by + j + 2 == 2    )) { um = -um; }
    // Fix north boundary for reconstruction of eta (corresponding to Ly)
    if ((bc_north_ == 1) && (by + j + 2 == ny_+2)) { up = -up; }
    
    // Reconstruct momentum along east
    const float up_east = up*east.x + vp*east.y;
    const float um_east = um*east.x + vm*east.y;
    
    // Reconstruct h
    const float hp = fmaxf(0.0f, eta_bar_p + H_face - ( Ly_p - dy_*coriolis_fp*up_east)/(2.0f*g_));
    const float hm = fmaxf(0.0f, eta_bar_m + H_face + ( Ly_m - dy_*coriolis_fm*um_east)/(2.0f*g_));

    // Our flux variables Q=(h, v, u)
    // Note that we swap u and v
    const float3 Qp = make_float3(hp, Rp.y, Rp.x);
    const float3 Qm = make_float3(hm, Rm.y, Rm.x);

    // Computed flux
    // Note that we swap back u and v
    const float3 flux = CDKLM16_flux(Qm, Qp, g_);
    return make_float3(flux.x, flux.z, flux.y);
}


__device__ 
void handleWallBC(
                const int& nx_, const int& ny_,
                const int& ti_, const int& tj_, 
                const int& tx_, const int& ty_, 
                const int& bc_north_, const int& bc_south_,
                const int& bc_east_, const int& bc_west_,
                float R[3][block_height+4][block_width+4]) {
    const int wall_bc = 1;

    const int i = tx_ + 2; //Skip local ghost cells, i.e., +2
    const int j = ty_ + 2;
        
    if (bc_north_ == wall_bc && tj_ == ny_+1) {
        R[0][j+1][i] =  R[0][j][i];
        R[1][j+1][i] =  R[1][j][i];
        R[2][j+1][i] = -R[2][j][i];

        R[0][j+2][i] =  R[0][j-1][i];
        R[1][j+2][i] =  R[1][j-1][i];
        R[2][j+2][i] = -R[2][j-1][i];
    }
    
    if (bc_south_ == wall_bc && tj_ == 2) {
        R[0][j-1][i] =  R[0][j][i];
        R[1][j-1][i] =  R[1][j][i];
        R[2][j-1][i] = -R[2][j][i];

        R[0][j-2][i] =  R[0][j+1][i];
        R[1][j-2][i] =  R[1][j+1][i];
        R[2][j-2][i] = -R[2][j+1][i];
    }
    
    if (bc_east_ == wall_bc && ti_ == nx_+1) {
        R[0][j][i+1] =  R[0][j][i];
        R[1][j][i+1] = -R[1][j][i];
        R[2][j][i+1] =  R[2][j][i];

        R[0][j][i+2] =  R[0][j][i-1];
        R[1][j][i+2] = -R[1][j][i-1];
        R[2][j][i+2] =  R[2][j][i-1];
    }
    
    if (bc_west_ == wall_bc && ti_ == 2) {
        R[0][j][i-1] =  R[0][j][i];
        R[1][j][i-1] = -R[1][j][i];
        R[2][j][i-1] =  R[2][j][i];

        R[0][j][i-2] =  R[0][j][i+1];
        R[1][j][i-2] = -R[1][j][i+1];
        R[2][j][i-2] =  R[2][j][i+1];
    }
}


/**
  * Uses a matrix stored as float 4
  * [x, y] * [u] = [x*u + y*v]
  * [z, w]   [v]   [z*u + w*v]
  * and multiply 
  */
__device__
inline float2 matMul(float4 M, float2 v) {
    return make_float2(M.x*v.x + M.y*v.y, M.z*v.x + M.w*v.y);
}

//texture<float, hipTextureType2D> angle_tex;

extern "C" {
__global__ void cdklm_swe_2D(
        const int nx_, const int ny_,
        const float dx_, const float dy_, const float dt_,
        const float g_,

        const float theta_,

        const float f_, //< Coriolis coefficient (f_ - beta_*y0)
        const float beta_, //< Coriolis force f_ + beta_*y

        const float r_, //< Bottom friction coefficient

        const int rk_order, // runge kutta order
        const int step_,    // runge kutta step

        //Input h^n
        float* eta0_ptr_, const int eta0_pitch_,
        float* hu0_ptr_, const int hu0_pitch_,
        float* hv0_ptr_, const int hv0_pitch_,

        //Output h^{n+1}
        float* eta1_ptr_, const int eta1_pitch_,
        float* hu1_ptr_, const int hu1_pitch_,
        float* hv1_ptr_, const int hv1_pitch_,

        //Bathymery
        float* Hi_ptr_, const int Hi_pitch_,
        float* Hm_ptr_, const int Hm_pitch_,
        float land_value_,

        //Wind stress parameters
        const float wind_stress_t_,

        // Boundary conditions (1: wall, 2: periodic, 3: open boundary (flow relaxation scheme))
        // Note: these are packed north, east, south, west boolean bits into an int
        const int boundary_conditions_) {
            
    //const float land_value_ = 1.0e20;


    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    //Index of block within domain
    const int bx = blockDim.x * blockIdx.x;
    const int by = blockDim.y * blockIdx.y;

    //Index of cell within domain
    const int ti = blockIdx.x * blockDim.x + threadIdx.x + 2; //Skip global ghost cells, i.e., +2
    const int tj = blockIdx.y * blockDim.y + threadIdx.y + 2;

    // Our physical variables
    // Input is [eta, hu, hv]
    // Will store [eta, u, v] (Note u and v are actually computed somewhat down in the code)
    __shared__ float R[3][block_height+4][block_width+4];

    // Our reconstruction variables
    //When computing flux along x-axis, we use
    //Qx = [u_x, v_x, K_x]
    //Then we reuse it as
    //Qx = [u_y, v_y, L_y]
    //to compute the y fluxes
    __shared__ float Qx[3][block_height+2][block_width+2];

    // Bathymetry
    // Need to find H on all faces for the cells in the block (block_height+1, block_width+1)
    // and for one face further out to adjust for the Kx and Ly slope outside of the block
    __shared__ float  Hi[block_height+3][block_width+3];

    // Get the angle towards north and create the matrices for the basis transformation
    // Discarding ghost cells and adding 0.5 to account for texture pixel values
    const float s = (ti-2.0f+0.5f) / (float) nx_;
    const float t = (tj-2.0f+0.5f) / (float) ny_;
    
    const float angle = tex2D(angle_tex, s, t);
    const float cos_a = cosf(angle);
    const float sin_a = sinf(angle);
    
    // B transforms from [x, y] to [n, e] (rotates by theta)
    // B = np.array([[cos(theta), -sin(theta)], [sin(theta), cos(theta)]])
    const float4 B = make_float4(cos_a, -sin_a, sin_a, cos_a);
    
    // BT transforms from [e, n] to [x, y] (rotates by -theta)
    // BT = np.array([[cos(theta), sin(theta)], [-sin(theta), cos(theta)]])
    const float4 BT = make_float4(cos_a, sin_a, -sin_a, cos_a);
    
    // North and east vector in xy-coordinate system
    // Given x and y-aligned vectors, simply compute the dot product, 
    // i.e., 
    // hu_north = north.x*hu + north.y*hv
    // hu_east = east.x*hu + east.y*hv
    const float2 north = matMul(BT, make_float2(0.0, 1.0));
    const float2 east = make_float2(north.y, -north.x);
    
    //Up vector in east-north coordinate system
    // Given n and e-aligned vectors, simply compute the dot product,
    // i.e., 
    // hu = right.x*hu_north + right.y*hu_east
    // hv = up.x*hu_north + up.y*hu_east
    const float2 up = matMul(B, make_float2(0.0, 1.0));
    const float2 right = make_float2(up.y, -up.x);


    // theta_ = 1.5f;

    //Read into shared memory
    for (int j=ty; j<block_height+4; j+=blockDim.y) {
        const int l = clamp(by + j, 0, ny_+3); // Out of bounds

        //Compute the pointer to current row in the arrays
        float* const eta_row = (float*) ((char*) eta0_ptr_ + eta0_pitch_*l);
        float* const hu_row = (float*) ((char*) hu0_ptr_ + hu0_pitch_*l);
        float* const hv_row = (float*) ((char*) hv0_ptr_ + hv0_pitch_*l);

        for (int i=tx; i<block_width+4; i+=blockDim.x) {
            const int k = clamp(bx + i, 0, nx_+3); // Out of bounds

            R[0][j][i] = eta_row[k];
            R[1][j][i] = hu_row[k];
            R[2][j][i] = hv_row[k];
        }
    }
    __syncthreads();
    

    // Read Hi into shared memory
    // Read intersections on all non-ghost cells
    for(int j=ty; j < block_height+3; j+=blockDim.y) {
        // Skip ghost cells and
        const int l = clamp(by+j+1, 1, ny_+4);
        float* const Hi_row = (float*) ((char*) Hi_ptr_ + Hi_pitch_*l);
        for(int i=tx; i < block_width+3; i+=blockDim.x) {
            const int k = clamp(bx+i+1, 1, nx_+4);

            Hi[j][i] = Hi_row[k];
            
            if (fabsf(Hi[j][i] - land_value_) < CDKLM_DRY_EPS) {
                Hi[j][i] = CDKLM_DRY_FLAG;
            }
        }
    }
    __syncthreads();
    
    
    
    // Compute Coriolis terms needed for fluxes etc.
    // The beta plane is defined relative to the lower corner of cell (2, 2),
    // which is the first internal cell that is not a ghost cell.
    //beta * (i*dx, j*dy)*(north.x, north.y)
    const float coriolis_f_lower   = f_ + beta_ * ((ti+0.5f-2.0f)*dx_*north.x + (tj-0.5f-2.0f)*dy_*north.y);
    const float coriolis_f_central = f_ + beta_ * ((ti+0.5f-2.0f)*dx_*north.x + (tj+0.5f-2.0f)*dy_*north.y);
    const float coriolis_f_upper   = f_ + beta_ * ((ti+0.5f-2.0f)*dx_*north.x + (tj+1.5f-2.0f)*dy_*north.y);
    const float coriolis_f_left    = f_ + beta_ * ((ti-0.5f-2.0f)*dx_*north.x + (tj+0.5f-2.0f)*dy_*north.y);
    const float coriolis_f_right   = f_ + beta_ * ((ti+1.5f-2.0f)*dx_*north.x + (tj+0.5f-2.0f)*dy_*north.y);


    //Fix boundary conditions
    //This must match code in CDKLM16.py:callKernel(...)
    const int bc_north = (boundary_conditions_ >> 24) & 0xFF;
    const int bc_south = (boundary_conditions_ >> 16) & 0xFF;
    const int bc_east = (boundary_conditions_ >> 8) & 0xFF;
    const int bc_west = (boundary_conditions_ >> 0) & 0xFF;
    
    if (boundary_conditions_ > 0) {
        // These boundary conditions are dealt with inside shared memory
        handleWallBC(nx_, ny_,
                ti, tj,
                tx, ty,
                bc_north, bc_south,
                bc_east, bc_west,
                R);
    }

    __syncthreads();
    
    // Compensate for one layer of ghost cells
    float Hm = 0.25f*(Hi[ty+1][tx+1] + Hi[ty+2][tx+1] + Hi[ty+1][tx+2] + Hi[ty+2][tx+2]);

    //Create our "steady state" reconstruction variables (u, v)
    // K and L are never stored, but computed where needed.
    for (int j=ty; j<block_height+4; j+=blockDim.y) {
        const int l = clamp(by+j, 0, ny_+3);
        float* const Hm_row = (float*) ((char*) Hm_ptr_ + Hm_pitch_*l);
        for (int i=tx; i<block_width+4; i+=blockDim.x) {
            const int k = clamp(bx+i, 0, nx_+3);

            // h = eta + H
            const float local_Hm = Hm_row[k];
            //const float local_Hm = 0.25f*(Hi[j][i] + Hi[j+1][i] + Hi[j][i+1] + Hi[j+1][i+1]);
            const float h = R[0][j][i] + local_Hm;
            
            //Check if this cell is actually dry (or land)
            //NOTE: This requires that all four corners of a cell are dry to be considered dry cell
            if (fabsf(local_Hm - land_value_) <= CDKLM_DRY_EPS) {
                R[0][j][i] = CDKLM_DRY_FLAG;
                R[1][j][i] = 0.0f;
                R[2][j][i] = 0.0f;
            }
            // Check if the cell is almost dry
            else if (h < KPSIMULATOR_DESING_EPS) {
                
                if (h <= KPSIMULATOR_DEPTH_CUTOFF) {
                    R[0][j][i] = -local_Hm + KPSIMULATOR_DEPTH_CUTOFF;
                    R[1][j][i] = 0.0f;
                    R[2][j][i] = 0.0f;
                }
                else {                
                    // Desingularizing u and v
                    //R[0][j][i] = h - local_Hm;
                    R[1][j][i] = desingularize(h, R[1][j][i], KPSIMULATOR_DESING_EPS); 
                    R[2][j][i] = desingularize(h, R[2][j][i], KPSIMULATOR_DESING_EPS); 
                }
            }
            // Wet cells
            else {
                R[1][j][i] /= h;
                R[2][j][i] /= h;
            }

            
        }
    }
    __syncthreads();

    // Store desingulized hu and hv
    //Skip local ghost cells, i.e., +2
    float hu = 0.0f;
    float hv = 0.0f;
    if ((R[0][ty + 2][tx + 2] + Hm) > KPSIMULATOR_DEPTH_CUTOFF) {
        hu = R[1][ty + 2][tx + 2]*(R[0][ty + 2][tx + 2] + Hm);
        hv = R[2][ty + 2][tx + 2]*(R[0][ty + 2][tx + 2] + Hm);
    }




    //Reconstruct slopes along x axis
    // Write result into shmem Qx = [u_x, v_x, K_x]
    // Qx is used as if its size was Qx[3][block_height][block_width + 2]
    for (int j=ty; j<block_height; j+=blockDim.y) {
        const int l = j + 2; //Skip ghost cells
        for (int i=tx; i<block_width+2; i+=blockDim.x) {
            const int k = i + 1;

            const float left_eta   = R[0][l][k-1];
            const float center_eta = R[0][l][k  ];
            const float right_eta  = R[0][l][k+1];

            const float left_u   = R[1][l][k-1];
            const float center_u = R[1][l][k  ];
            const float right_u  = R[1][l][k+1];
        
            float left_v   = R[2][l][k-1];
            float center_v = R[2][l][k  ];
            float right_v  = R[2][l][k+1];
            
            Qx[0][j][i] = minmodSlope(left_u, center_u, right_u, theta_);
            Qx[1][j][i] = minmodSlope(left_v, center_v, right_v, theta_);
            
            // Enforce wall boundary conditions for Kx:
            int global_thread_id_x = bx + i + 1; // index including ghost cells'
            // Western BC
            if (bc_west == 1) {
                if (global_thread_id_x < 3    ) { left_v   = -left_v;   }
                if (global_thread_id_x < 2    ) { center_v = -center_v; }
            }
            // Eastern BC
            if (bc_east == 1) {
                if (global_thread_id_x > nx_  ) { right_v  = -right_v;  }
                if (global_thread_id_x > nx_+1) { center_v = -center_v; }
            }
            
            // Get north vector for thread (bx + k, by +l)
            const float2 local_north = getNorth(bx+k, by+l, nx_, ny_);
            
            const float left_coriolis_f   = f_ + beta_ * ((bx + k - 0.5f-2.0f)*dx_*local_north.x + (by + l + 0.5f-2.0f)*dy_*local_north.y);
            const float center_coriolis_f = f_ + beta_ * ((bx + k + 0.5f-2.0f)*dx_*local_north.x + (by + l + 0.5f-2.0f)*dy_*local_north.y);
            const float right_coriolis_f  = f_ + beta_ * ((bx + k + 1.5f-2.0f)*dx_*local_north.x + (by + l + 0.5f-2.0f)*dy_*local_north.y);
            
            const float left_fv  = (local_north.x*left_u + local_north.y*left_v)*left_coriolis_f;
            const float center_fv = (local_north.x*center_u + local_north.y*center_v)*center_coriolis_f;
            const float right_fv  = (local_north.x*right_u + local_north.y*right_v)*right_coriolis_f;
            
            const float V_constant = dx_/(2.0f*g_);

            // Qx[2] = Kx, which we need to find differently than ux and vx
            const float backward = theta_*g_*(center_eta - left_eta   - V_constant*(center_fv + left_fv ) );
            const float central  =   0.5f*g_*(right_eta  - left_eta   - V_constant*(right_fv + 2*center_fv + left_fv) );
            const float forward  = theta_*g_*(right_eta  - center_eta - V_constant*(center_fv + right_fv) );

            // Qx[2] is really dx*Kx
            Qx[2][j][i] = minmodRaw(backward, central, forward);

        }
    }
    __syncthreads();
        
    // Adjust K_x slopes to avoid negative h = eta + H
    // Need K_x (Qx[2]), coriolis parameter (f, beta), eta (R[0]), v (R[2]), H (Hi), g, dx
    adjustSlopes_x(bx, by, nx_, ny_, dx_, dy_,
                   R, Qx, Hi,
                   g_, f_, beta_, 
                   bc_east, bc_west);
    __syncthreads();
    
    // Compute flux along x axis
    float3 flux_diff = (  
            computeFFaceFlux(
                tx+1, ty, bx, nx_, 
                R, Qx, Hi,
                g_, coriolis_f_central, coriolis_f_right, 
                dx_, 
                bc_north, bc_south, 
                north)
            - 
            computeFFaceFlux(
                tx , ty, bx, nx_, 
                R, Qx, Hi,
                g_, coriolis_f_left, coriolis_f_central, 
                dx_, 
                bc_north, bc_south, 
                north)) / dx_;
    __syncthreads();
    
    // Reconstruct eta_west, eta_east for use in bathymetry source term
    const float eta_west = R[0][ty+2][tx+2] - (Qx[2][ty][tx+1] + dx_*coriolis_f_central*R[2][ty+2][tx+2])/(2.0f*g_);
    const float eta_east = R[0][ty+2][tx+2] + (Qx[2][ty][tx+1] + dx_*coriolis_f_central*R[2][ty+2][tx+2])/(2.0f*g_);
    
    __syncthreads();
    
    //Reconstruct slopes along y axis
    // Write result into shmem Qx = [u_y, v_y, L_y]
    // Qx is now used as if its size was Qx[3][block_height+2][block_width]

    for (int j=ty; j<block_height+2; j+=blockDim.y) {
        const int l = j + 1;
        for (int i=tx; i<block_width; i+=blockDim.x) {
            const int k = i + 2; //Skip ghost cells
            // Qy[2] = Ly, which we need to find differently than uy and vy
            const float lower_eta  = R[0][l-1][k];
            const float center_eta = R[0][l  ][k];
            const float upper_eta  = R[0][l+1][k];

            float lower_u  = R[1][l-1][k];
            float center_u = R[1][l  ][k];
            float upper_u  = R[1][l+1][k];

            const float lower_v  = R[2][l-1][k];
            const float center_v = R[2][l  ][k];
            const float upper_v  = R[2][l+1][k];
            
            Qx[0][j][i] = minmodSlope(lower_u, center_u, upper_u, theta_);
            Qx[1][j][i] = minmodSlope(lower_v, center_v, upper_v, theta_);

            // Enforce wall boundary conditions for Ly
            int global_thread_id_y = by + j + 1; // index including ghost cells
            // southern BC
            if (bc_south == 1) {
                if (global_thread_id_y < 3    ) { lower_u  = -lower_u;  }
                if (global_thread_id_y < 2    ) { center_u = -center_u; }
            }
            // northern BC
            if (bc_north == 1) {
                if (global_thread_id_y > ny_  ) { upper_u  = -upper_u;  }
                if (global_thread_id_y > ny_+1) { center_u = -center_u; }
            }
            
            // Get north and east vectors for thread (bx + k, by +l)
            const float2 local_north = getNorth(bx+k, by+l, nx_, ny_);
            const float2 local_east = getEast(bx+k, by+l, nx_, ny_);
            
            const float lower_coriolis_f  = f_ + beta_ * ((bx + k + 0.5f-2.0f)*dx_*local_north.x + (by + l - 0.5f-2.0f)*dy_*local_north.y);
            const float center_coriolis_f = f_ + beta_ * ((bx + k + 0.5f-2.0f)*dx_*local_north.x + (by + l + 0.5f-2.0f)*dy_*local_north.y);
            const float upper_coriolis_f  = f_ + beta_ * ((bx + k + 0.5f-2.0f)*dx_*local_north.x + (by + l + 1.5f-2.0f)*dy_*local_north.y);

            const float lower_fu  = (local_east.x*lower_u  + local_east.y*lower_v )*lower_coriolis_f;
            const float center_fu = (local_east.x*center_u + local_east.y*center_v)*center_coriolis_f;
            const float upper_fu  = (local_east.x*upper_u  + local_east.y*upper_v )*upper_coriolis_f;

            const float U_constant = dy_/(2.0f*g_);

            const float backward = theta_*g_*(center_eta - lower_eta  + U_constant*(center_fu + lower_fu ) );
            const float central  =   0.5f*g_*(upper_eta  - lower_eta  + U_constant*(upper_fu + 2*center_fu + lower_fu) );
            const float forward  = theta_*g_*(upper_eta  - center_eta + U_constant*(center_fu + upper_fu) );

            // Qy[2] is really dy*Ly
            Qx[2][j][i] = minmodRaw(backward, central, forward);
        }
    }
    __syncthreads();

    // Adjust L_y slopes to avoid negative h = eta + H
    // Need L_x (Qx[2]), coriolis parameter (f, beta), eta (R[0]), u (R[1]), H (Hi), g, dx
    adjustSlopes_y(bx, by, nx_, ny_, dx_, dy_,
                   R, Qx, Hi,
                   g_, f_, beta_, 
                   bc_north, bc_south);
    __syncthreads();
    
    //Compute fluxes along the y axis
    flux_diff = flux_diff + 
        (computeGFaceFlux(
            tx, ty+1, by, ny_, 
            R, Qx, Hi, 
            g_, coriolis_f_central, coriolis_f_upper, 
            dy_, 
            bc_east, bc_west, 
            east)
        - 
        computeGFaceFlux(
            tx, ty, by, ny_, 
            R, Qx, Hi, 
            g_, coriolis_f_lower, coriolis_f_central, 
            dy_, 
            bc_east, bc_west, 
            east)) / dy_;
    __syncthreads();

    // Reconstruct eta_north, eta_south for use in bathymetry source term
    const float eta_south = R[0][ty+2][tx+2] - (Qx[2][ty+1][tx] - dy_*coriolis_f_central*R[1][ty+2][tx+2])/(2.0f*g_);
    const float eta_north = R[0][ty+2][tx+2] + (Qx[2][ty+1][tx] - dy_*coriolis_f_central*R[1][ty+2][tx+2])/(2.0f*g_);
    __syncthreads();
    
    //Sum fluxes and advance in time for all internal cells
    if (ti > 1 && ti < nx_+2 && tj > 1 && tj < ny_+2) {
        //Skip local ghost cells, i.e., +2
        const int i = tx + 2; 
        const int j = ty + 2;
        
        // Skip local ghost cells for Hi
        const int H_i = tx + 1;
        const int H_j = ty + 1;

        // Source terms (wind, coriolis, bathymetry)
        float st1 = 0.0f;
        float st2 = 0.0f;
        
        const float h = R[0][j][i] + Hm;
        //If wet cell
        if (h >= KPSIMULATOR_DEPTH_CUTOFF) {
            // If not land
            if (R[0][j][i] != CDKLM_DRY_FLAG) {
                // Wind
                const float X = windStressX(wind_stress_t_, ti+0.5, tj+0.5, nx_, ny_);
                const float Y = windStressY(wind_stress_t_, ti+0.5, tj+0.5, nx_, ny_);

                // Bottom topography source terms!
                // -g*(eta + H)*(-1)*dH/dx   * dx
                const float RHxp = 0.5f*( Hi[H_j  ][H_i+1] + Hi[H_j+1][H_i+1] );
                const float RHxm = 0.5f*( Hi[H_j  ][H_i  ] + Hi[H_j+1][H_i  ] );
                const float RHyp = 0.5f*( Hi[H_j+1][H_i  ] + Hi[H_j+1][H_i+1] );
                const float RHym = 0.5f*( Hi[H_j  ][H_i  ] + Hi[H_j  ][H_i+1] );
                
                float H_x = RHxp - RHxm;
                float H_y = RHyp - RHym;
                
                const float eta_sn = 0.5f*(eta_north + eta_south);
                const float eta_we = 0.5f*(eta_west  + eta_east);

                // TODO: We might want to use the mean of the reconstructed eta's at the faces here, instead of R[0]...
                //const float bathymetry1 = g_*(R[0][j][i] + Hm)*H_x;
                //const float bathymetry2 = g_*(R[0][j][i] + Hm)*H_y;
                const float bathymetry1 = g_*(eta_we + Hm)*H_x;
                const float bathymetry2 = g_*(eta_sn + Hm)*H_y;
                
                //Find north-going and east-going coriolis force
                const float hu_east =  coriolis_f_central*(hu*east.x + hv*east.y);
                const float hv_north = coriolis_f_central*(hu*north.x + hv*north.y);
                
                //Convert back to xy coordinate system
                const float hu_cor = right.x*hu_east + right.y*hv_north;
                const float hv_cor = up.x*hu_east + up.y*hv_north;

                // Total source terms
                st1 = X + hv_cor + bathymetry1/dx_;
                st2 = Y - hu_cor + bathymetry2/dy_;
            }
        }

        
        const float L1  = - flux_diff.x;
        const float L2  = - flux_diff.y + st1;
        const float L3  = - flux_diff.z + st2;

        float* const eta_row = (float*) ((char*) eta1_ptr_ + eta1_pitch_*tj);
        float* const hu_row  = (float*) ((char*) hu1_ptr_  +  hu1_pitch_*tj);
        float* const hv_row  = (float*) ((char*) hv1_ptr_  +  hv1_pitch_*tj);

        float updated_eta;
        float updated_hu;
        float updated_hv;
        
        if (rk_order < 3) {

#ifdef use_linear_friction
            const float C = 2.0f*r_*dt_/(R[0][j][i] + Hm);
#else
            float C = 0.0;
            if (r_ > 0.0) {
                if (h < KPSIMULATOR_DESING_EPS) {
                    const float u = desingularize(h, hu, KPSIMULATOR_DESING_EPS);
                    const float v = desingularize(h, hv, KPSIMULATOR_DESING_EPS);
                    C = dt_*r_*sqrt(u*u+v*v)/h;
                }
                else {
                    const float u = hu/h;
                    const float v = hv/h;
                    C = dt_*r_*sqrt(u*u+v*v)/h;
                }
            }
#endif
            
            if  (step_ == 0) {
                //First step of RK2 ODE integrator

                updated_eta =  R[0][j][i] + dt_*L1;
                updated_hu  = (hu + dt_*L2) / (1.0f + C);
                updated_hv  = (hv + dt_*L3) / (1.0f + C);
            }
            else if (step_ == 1) {
                //Second step of RK2 ODE integrator

                //First read Q^n
                const float eta_a = eta_row[ti];
                const float hu_a  =  hu_row[ti];
                const float hv_a  =  hv_row[ti];

                //Compute Q^n+1
                const float eta_b = 0.5f*(eta_a + (R[0][j][i] + dt_*L1));
                const float hu_b  = 0.5f*( hu_a + (hu + dt_*L2));
                const float hv_b  = 0.5f*( hv_a + (hv + dt_*L3));


                //Write to main memory
                updated_eta = eta_b;
                updated_hu  =  hu_b / (1.0f + 0.5f*C);
                updated_hv  =  hv_b / (1.0f + 0.5f*C);

            }
        }


        else if (rk_order == 3) {
            // Third order Runge Kutta - only valid if r_ = 0.0 (no friction)

            if (step_ == 0) {
                //First step of RK3 ODE integrator
                // q^(1) = q^n + dt*L(q^n)

                updated_eta =  R[0][j][i] + dt_*L1;
                updated_hu  = (hu + dt_*L2);
                updated_hv  = (hv + dt_*L3);

            } else if (step_ == 1) {
                // Second step of RK3 ODE integrator
                // Q^(2) = 3/4 Q^n + 1/4 ( Q^(1) + dt*L(Q^(1)) )
                // Q^n is here in h1, but will be used in next iteration as well --> write to h0

                // First read Q^n:
                const float eta_a = eta_row[ti];
                const float hu_a  =  hu_row[ti];
                const float hv_a  =  hv_row[ti];

                // Compute Q^(2):
                const float eta_b = 0.75f*eta_a + 0.25f*(R[0][j][i] + dt_*L1);
                const float hu_b  = 0.75f* hu_a + 0.25f*(hu + dt_*L2);
                const float hv_b  = 0.75f* hv_a + 0.25f*(hv + dt_*L3);

                // Write output to the input buffer:
                updated_eta = eta_b;
                updated_hu  =  hu_b;
                updated_hv  =  hv_b;

            } else if (step_ == 2) {
                // Third step of RK3 ODE integrator
                // Q^n+1 = 1/3 Q^n + 2/3 (Q^(2) + dt*L(Q^(2))

                // First read Q^n:
                const float eta_a = eta_row[ti];
                const float hu_a  =  hu_row[ti];
                const float hv_a  =  hv_row[ti];

                // Compute Q^n+1:
                const float eta_b = (eta_a + 2.0f*(R[0][j][i] + dt_*L1)) / 3.0f;
                const float hu_b  = ( hu_a + 2.0f*(hu + dt_*L2)) / 3.0f;
                const float hv_b  = ( hv_a + 2.0f*(hv + dt_*L3)) / 3.0f;

                //Write to main memory
                updated_eta = eta_b;
                updated_hu  =  hu_b;
                updated_hv  =  hv_b;
            }
        }
    

        const float updated_h = updated_eta + Hm;
        if ((updated_h <= KPSIMULATOR_DEPTH_CUTOFF) ) { 
            updated_eta = -Hm + KPSIMULATOR_DEPTH_CUTOFF;
            updated_hu  = 0.0f;
            updated_hv  = 0.0f;
        }

        if ( (rk_order == 3) && (step_ == 1) ) {
            float* const eta_out_row = (float*) ((char*) eta0_ptr_ + eta0_pitch_*tj);
            float* const hu_out_row  = (float*) ((char*)  hu0_ptr_ +  hu0_pitch_*tj);
            float* const hv_out_row  = (float*) ((char*)  hv0_ptr_ +  hv0_pitch_*tj);

            eta_out_row[ti] = fmaxf(-Hm + KPSIMULATOR_DEPTH_CUTOFF, updated_eta);
            hu_out_row[ti]  = updated_hu;
            hv_out_row[ti]  = updated_hv;
        } else {
            eta_row[ti] = fmaxf(-Hm + KPSIMULATOR_DEPTH_CUTOFF, updated_eta);
            hu_row[ti]  = updated_hu;
            hv_row[ti]  = updated_hv;
        }
        
        
    }
}

}

